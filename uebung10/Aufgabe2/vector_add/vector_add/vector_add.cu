#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"

void cudaVectorAdd(float *a, float *b, float *c, int numElements, int repetitions, bool warmup) {
	float hostToDevice = 0;
	float compute = 0;
	float deviceToHost = 0;
	clock_t start = clock();

	for (int i = 0; i < repetitions; i++)
	{
		// TODO: Implement a parallel vector addition on CUDA
	}

	float total = float(clock() - start) / (CLOCKS_PER_SEC * repetitions);

	if (!warmup)
	{
		printf("CUDA: %.3lf seconds\n", total);
		printf("CUDA: Copy input to device: %.3lf seconds\n", hostToDevice / (1000 * repetitions));
		printf("CUDA: Compute time: %.3lf seconds\n", compute / (1000 * repetitions));
		printf("CUDA: Copy output to host: %.3lf seconds\n", deviceToHost / (1000 * repetitions));
	}
}

void fillRandomArray(float *a, int numElements) {
	for (int i = 0; i < numElements; i++) {
		a[i] = rand() / (float)RAND_MAX;
	}
}

void verifyResults(float *a, float *b, float *c, int numElements) {
	for (int i = 0; i < numElements; i++) {
        if (fabs(a[i] + b[i] - c[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
}

void sequentialVectorAdd(float *a, float *b, float *c, int numElements) {
	clock_t start = clock();

	for (int i = 0; i < numElements; i++) {
		c[i] = a[i] + b[i];
	}

	float diff = float(clock() - start) / CLOCKS_PER_SEC;
	printf("Sequential: %.3lf seconds\n", diff);
}

int main() {
	int N = 10000000;
	size_t size = N * sizeof(float);

	float *h_a = (float *)malloc(size);
	handleAllocationError(h_a);
	fillRandomArray(h_a, N);
	
	float *h_b = (float *)malloc(size);
	handleAllocationError(h_b);
	fillRandomArray(h_b, N);
	
	float *h_c = (float *)malloc(size);
	handleAllocationError(h_c);

	cudaVectorAdd(h_a, h_b, h_c, N, 10, true);
	verifyResults(h_a, h_b, h_c, N);
	cudaVectorAdd(h_a, h_b, h_c, N, 100, false);

	sequentialVectorAdd(h_a, h_b, h_c, N);

	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}
