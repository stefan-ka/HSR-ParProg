#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"

// See CUDA specification
int numberOfCoresPerMP(int majorVersion, int minorVersion) {
    if (majorVersion == 1) {
        return 8;
    }
    else if (majorVersion == 2 && minorVersion == 0) {
        return 32;
    }
    else if (majorVersion == 2 && minorVersion == 1) {
        return 48;
    }
    else if (majorVersion == 3) {
        return 192;
    }
    else if (majorVersion == 5) {
        return 128;
    }
    else {
        return 0; // unsupported version
    }
}

int main() {
    int deviceCount;
    handleCudaError(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        printf("No CUDA devices");
    }
    for (int deviceNumber = 0; deviceNumber < deviceCount; deviceNumber++) {
        hipSetDevice(deviceNumber);

        hipDeviceProp_t properties;
        handleCudaError(hipGetDeviceProperties(&properties, deviceNumber));
        printf("CUDA Device: %d %s\n", deviceNumber, properties.name);

        int driverVersion, runtimeVersion;
        handleCudaError(hipDriverGetVersion(&driverVersion));
        handleCudaError(hipRuntimeGetVersion(&runtimeVersion));
        printf("  CUDA driver version			%i.%i\n", driverVersion / 1000, (driverVersion % 100) / 10);
        printf("  CUDA runtime version			%i.%i\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
        printf("  CUDA capability major / minor		%i.%i\n", properties.major, properties.minor);
        printf("  Global memory				%i MB\n", (int)(properties.totalGlobalMem / 1024 / 1024));
        printf("  Cores per multiprocessor		%i\n", numberOfCoresPerMP(properties.major, properties.minor));
        printf("  Multiprocessor count			%i\n", properties.multiProcessorCount);
        printf("  Number of cores			%i\n", numberOfCoresPerMP(properties.major, properties.minor) * properties.multiProcessorCount);
        printf("  Clock rate				%i MHz\n", properties.clockRate / 1000);
        printf("  Memory clock rate			%i MHz\n", properties.memoryClockRate / 1000);
        printf("  Memory bus width			%i bits\n", properties.memoryBusWidth);
        printf("  L2 cache size				%lu bytes\n", properties.l2CacheSize);
        printf("  Constant memory			%lu bytes\n", properties.totalConstMem);
        printf("  Shared memory per block		%lu bytes\n", properties.sharedMemPerBlock);
        printf("  Registers per block			%i\n", properties.regsPerBlock);
        printf("  Warp size				%i\n", properties.warpSize);
        printf("  Maximum threads per multiprocessor	%i\n", properties.maxThreadsPerMultiProcessor);
        printf("  Maximum threads per block		%i\n", properties.maxThreadsPerBlock);
        printf("  Max thread dimension per block	(%i, %i, %i)\n", properties.maxThreadsDim[0], properties.maxThreadsDim[1], properties.maxThreadsDim[2]);
        printf("  Max grid size				(%i, %i, %i)\n", properties.maxGridSize[0], properties.maxGridSize[1], properties.maxGridSize[2]);
    }
    return 0;
}