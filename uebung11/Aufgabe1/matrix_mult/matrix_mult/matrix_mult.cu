#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"

// C = A * B

#define C_ROWS 1000
#define C_COLS 2000
#define A_COLS 3000

#define A_ROWS C_ROWS 
#define B_ROWS A_COLS 
#define B_COLS C_COLS 

void cudaMatrixMult(float *A, float *B, float *C, int repetitions, bool warmup) {
	clock_t start = clock();

	for (int i = 0; i < repetitions; i++)
	{
		// TODO: Implement parallel matrix multiplication on CUDA (simple version)
	}
	if (!warmup)
	{
		float diff = float(clock() - start) / (CLOCKS_PER_SEC * repetitions);
		printf("CUDA: %.3lf seconds\n", diff);
	}
}

void fillRandomArray(float *A, int numElements) {
	for (int i = 0; i < numElements; i++) {
		A[i] = rand() / (float)RAND_MAX;
	}
}

void verifyResults(float *A, float *B, float *C) {
	for (int row = 0; row < C_ROWS; row++) {
		for (int col = 0; col < C_COLS; col++) {
			float sum = 0.0;
			for (int k = 0; k < A_COLS; k++) {
				sum += A[row * A_COLS + k] * B[k * B_COLS + col];
			}
			if (fabs(C[row * C_COLS + col] - sum) > 1e-3) {
				fprintf(stderr, "Result verification failed at element %d: %f vs. %f!\n", row, C[row * C_COLS + col], sum);
				exit(EXIT_FAILURE);
			}
		}
	}
}

void sequentialMatrixMult(float *A, float *B, float *C) {
	clock_t start = clock();

	for (int row = 0; row < C_ROWS; row++) {
		for (int col = 0; col < C_COLS; col++) {
			float sum = 0.0;
			for (int k = 0; k < A_COLS; k++) {
				sum += A[row * A_COLS + k] * B[k * B_COLS + col];
			}
			C[row * C_COLS + col] = sum;
		}
	}

	float diff = float(clock() - start) / CLOCKS_PER_SEC;
	printf("Sequential: %.3lf seconds\n", diff);
}

int main() {
	int nofElemA = A_ROWS * A_COLS;
	float *h_A = (float *)malloc(nofElemA * sizeof(float));
	handleAllocationError(h_A);
	fillRandomArray(h_A, nofElemA);

	int nofElemB = B_ROWS * B_COLS;
	float *h_B = (float *)malloc(nofElemB * sizeof(float));
	handleAllocationError(h_B);
	fillRandomArray(h_B, nofElemB);
	
	int nofElemC = C_ROWS * C_COLS;
	float *h_C = (float *)malloc(nofElemC * sizeof(float));
	handleAllocationError(h_C);

	cudaMatrixMult(h_A, h_B, h_C, 2, true);
	verifyResults(h_A, h_B, h_C);
	cudaMatrixMult(h_A, h_B, h_C, 4, false);

	sequentialMatrixMult(h_A, h_B, h_C);

	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
